#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <ATen/native/cuda/Resize.cuh>

namespace at { namespace native {

Tensor& resize_cuda_(Tensor& self, IntArrayRef size) {
#ifdef BUILD_NAMEDTENSOR
  if (self.has_names()) {
    return resize_named_tensor_(self, size);
  }
#endif
  auto* self_ = self.unsafeGetTensorImpl();
  resize_impl_cuda_(self_, size, /*strides=*/c10::nullopt);
  self_->maybe_zero_dim(size.size() == 0);
  return self;
}

Tensor& resize_as_cuda_(Tensor& self, const Tensor& the_template) {
  Tensor& result = resize_cuda_(self, the_template.sizes());
#ifdef BUILD_NAMEDTENSOR
  namedinference::propagate_names(result, the_template);
#endif
  return result;
}

}}
